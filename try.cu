#include "hip/hip_runtime.h"
#include "try.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void hi() {
    hiprandState_t state;
    hiprand_init((unsigned long) clock(), blockIdx.x, 0, &state);
    printf("random1:%u\n", hiprand(&state));
    hiprand_init((unsigned long) clock(), blockIdx.x, 0, &state);
    printf("random2:%u\n", hiprand(&state));
}
