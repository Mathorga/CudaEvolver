#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "hpc.h"
#include "CUDAPopulation.h"
#include "CUDAPathGenome.h"
#include "try.h"

// Translates bidimensional indexes to a monodimensional one.
// |i| is the column index.
// |j| is the row index.
// |n| is the number of columns (length of the rows).
#define IDX(i, j, n) ((i) * (n) + (j))

typedef unsigned char cell_t;

enum CellContent {
    EMPTY = 0,
    CHECK = 1,
    PATH = 2
};

// Code taken from Rosettacode:
// https://rosettacode.org/wiki/Bitmap/Bresenham%27s_line_algorithm#C.2B.2B
// Implementing Bresenham’s line drawing algorithm.
void drawLine(cell_t *field, int n, int x0, int y0, int x1, int y1) {
    int x = x0;
    int y = y0;

    int dx = abs(x1 - x0), sx = x0 < x1 ? 1 : -1;
    int dy = abs(y1 - y0), sy = y0 < y1 ? 1 : -1;
    int err = (dx > dy ? dx : -dy) / 2;
    int e2;

    for(;;){
        if (!(x == x0 && y == y0) && !(x == x1 && y == y1)) {
            field[IDX(x, y, n)] = PATH;
        }
        if (x == x1 && y == y1) {
            break;
        }
        e2 = err;
        if (e2 > -dx) {
            err -= dy;
            x += sx;
        }
        if (e2 < dy) {
            err += dx;
            y += sy;
        }
    }
}

void dump(const cell_t *field, const CUDAPathGenome::_Point2D *path, unsigned int n, unsigned int checksNum, const char *filename) {
    cell_t *fieldCopy = (cell_t *) malloc(n * n * sizeof(cell_t));
    for (unsigned int x = 0; x < n; x++) {
        for (unsigned int y = 0; y < n; y++) {
            fieldCopy[IDX(x, y, n)] = field[IDX(x, y, n)];
        }
    }

    if (path != NULL) {
        for (unsigned int i = 0; i < checksNum; i++) {
            drawLine(fieldCopy, n, path[i].x, path[i].y, path[(i + 1) % checksNum].x, path[(i + 1) % checksNum].y);
        }
    }
    FILE *out = fopen(filename, "w");
    if (out == NULL) {
        printf("Cannot create \"%s\"\n", filename);
        abort();
    }
    fprintf(out, "P6\n");
    fprintf(out, "%d %d\n", n, n);
    fprintf(out, "255\n");
    for (unsigned int x = 0; x < n; x++) {
        for (unsigned int y = 0; y < n; y++) {
            if (field[IDX((x + 1) % n, y, n)] == CHECK ||
                field[IDX(x, (y + 1) % n, n)] == CHECK ||
                field[IDX((x + 1) % n, (y + 1) % n, n)] == CHECK ||
                field[IDX((x - 1 + n) % n, y, n)] == CHECK ||
                field[IDX(x, (y - 1 + n) % n, n)] == CHECK ||
                field[IDX((x - 1 + n) % n, (y - 1 + n) % n, n)] == CHECK ||
                field[IDX((x + 1) %n, (y - 1 + n) % n, n)] == CHECK ||
                field[IDX((x - 1 + n) %n, (y + 1) % n, n)] == CHECK) {
                fprintf(out, "%c%c%c", 255, 30, 30);
            } else if (field[IDX(x, y, n)] == CHECK) {
                fprintf(out, "%c%c%c", 20, 20, 0);
            } else if (fieldCopy[IDX(x, y, n)] == EMPTY) {
                fprintf(out, "%c%c%c", 20, 20, 20);
            } else if (fieldCopy[IDX(x, y, n)] == PATH) {
                fprintf(out, "%c%c%c", 250, 175, 53);
            } else {
                printf("Unknown cell state (%d) of cell %d-%d", fieldCopy[IDX(x, y, n)], x, y);
                abort();
            }
        }
    }
    fclose(out);
}


int main(int argc, char const *argv[]) {
    // hi<<<2, 10>>>();
    // hipDeviceSynchronize();

    unsigned int fieldSize = 500;
    unsigned int checksNumber = 15;
    unsigned int popSize = 50;
    unsigned int genNumber = 1000;
    float mutRate = 0.1;
    float crossRate = 1;

    char fileName[200];
    double startTime = 0.0;
    double endTime = 0.0;
    cell_t *field;
    CUDAPathGenome::_Point2D *checks;

    if (argc > 7) {
        printf("Usage: %s [fieldSize [checksNumber [popSize [genNumber [mutRate [crossRate]]]]]]\n", argv[0]);
        return -1;
    }
    if (argc > 1) {
        fieldSize = atoi(argv[1]);
    }
    if (argc > 2) {
        checksNumber = atoi(argv[2]);
    }
    if (argc > 3) {
        popSize = atoi(argv[3]);
    }
    if (argc > 4) {
        genNumber = atoi(argv[4]);
    }
    if (argc > 5) {
        mutRate = atof(argv[5]);
    }
    if (argc > 6) {
        crossRate = atof(argv[6]);
    }

    // Create a field of checks.
    field = (cell_t *) malloc(fieldSize * fieldSize * sizeof(cell_t));
    checks = (CUDAPathGenome::_Point2D *) malloc(checksNumber * sizeof(CUDAPathGenome::_Point2D));

    for (unsigned int i = 0; i < fieldSize * fieldSize; i++) {
        field[i] = EMPTY;
    }

    srand(time(NULL));
    for (unsigned int i = 0; i < checksNumber; i++) {
        checks[i].x = (rand() % fieldSize);
        checks[i].y = (rand() % fieldSize);
        checks[i].id = i;
        field[IDX(checks[i].x, checks[i].y, fieldSize)] = true;
    }

    dump(field, NULL, fieldSize, checksNumber, "field.ppm");

    std::cout << "Field:\n";
    for (unsigned int i = 0; i < checksNumber; i++) {
        std::cout << "x:" << checks[i].x << "\ty:" << checks[i].y << "\n";
    }

    CUDAGenome *genome;
    hipMalloc(&genome, sizeof(CUDAGenome *));
    createCUDAPathGenome<<<1, 1>>>(&genome, checks, checksNumber);

    CUDAPopulation *population = new CUDAPopulation(popSize, genNumber, new CUDAPathGenome(checks, checksNumber), CUDAPopulation::MINIMIZE);
    population->initialize();

    // First option.
    dim3 nChecks(checksNumber);
    evolve(population, nChecks);

    // Second option.
    // CUDAPopulation *d_pop;
    // hipMalloc(&d_pop, sizeof(CUDAPopulation *));
    // hipMemcpy(d_pop, population, sizeof(CUDAPopulation *) ,hipMemcpyHostToDevice);
    // dim3 gridSize(popSize);
    // dim3 blockSize(checksNumber);
    // for (unsigned int i = 0; i < population->getSize(); i++) {
    //     evaluate<<<>>>(d_pop);
    //     hipDeviceSynchronize();
    //     step<<<>>>(d_pop);
    //     hipDeviceSynchronize();
    // }
    // hipMemcpy(population, d_pop, sizeof(CUDAPopulation *) ,hipMemcpyDeviceToHost);

    return 0;
}
