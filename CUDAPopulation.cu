#include "hip/hip_runtime.h"
#include "CUDAPopulation.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

void evolve(CUDAPopulation *pop, dim3 genomeSize) {
    dim3 popSize(pop->getSize());
    printf("Population size:%u\n", pop->getSize());

    // Copy the population on the device.
    CUDAPopulation *d_pop;
    hipMalloc(&d_pop, sizeof(CUDAPopulation));
    hipMemcpy(d_pop, pop, sizeof(CUDAPopulation), hipMemcpyHostToDevice);
    printf("Copied the population on the device\n");

    CUDAGenome **d_individuals;
    d_individuals = (CUDAGenome **) malloc(pop->getSize() * sizeof(CUDAGenome *));
    for (unsigned int i = 0; i < pop->getSize(); i++) {
        pop->individuals[0]->allocateCopySingle(&(d_individuals[i]), &(pop->individuals[i]), hipMemcpyHostToDevice);
    }
    printf("Copied the individuals on the device\n");

    CUDAGenome **tmpD_individuals;
    hipMalloc(&tmpD_individuals, pop->getSize() * sizeof(CUDAGenome *));
    hipMemcpy(tmpD_individuals, d_individuals, pop->getSize() * sizeof(CUDAGenome *), hipMemcpyHostToDevice);

    hipMemcpy(&(d_pop->individuals), &tmpD_individuals, sizeof(CUDAGenome **), hipMemcpyHostToDevice);


    // Evolve.
    printf("Starting evolution loop\n");
    for (unsigned int i = 0; i < pop->getGenNumber(); i++) {
        evaluate<<<popSize, genomeSize>>>(d_pop);
        hipDeviceSynchronize();
        step<<<popSize, genomeSize>>>(d_pop);
        hipDeviceSynchronize();
    }

    // TODO Copy the population back to the host.
    hipMemcpy(pop, d_pop, sizeof(CUDAPopulation *), hipMemcpyDeviceToHost);
}

__global__ void evaluate(CUDAPopulation *pop) {
    if (threadIdx.x == 0) {
        printf("Started evaluation of individual %d\n", blockIdx.x);
        printf("Address:%p\n", pop->individuals[blockIdx.x]);
        printf("Size:%u\n", pop->individuals[blockIdx.x]->getXSize());
        printf("checksNumber:%d\n", ((CUDAPathGenome *) (pop->individuals[blockIdx.x]))->getChecksNum());
    }
    pop->individuals[blockIdx.x]->evaluate();
}

__global__ void step(CUDAPopulation *pop) {
    pop->step();
}

CUDAPopulation::CUDAPopulation(unsigned int popSize, unsigned int genNum, CUDAGenome *genome, Objective obj) {
    printf("Starting creation\n");
    genNumber = genNum;
    currentGen = 0;
    initialized = false;
    size = popSize;
    individuals = (CUDAGenome **) malloc(size * sizeof(CUDAGenome *));
    printf("Allocated individuals on host\n");

    // genome->allocateIndividuals(d_individuals, size);
    // hipMalloc(&d_individuals, size * sizeof(CUDAGenome *));

    printf("Allocated individuals' pointer on device\n");
    for (unsigned int i = 0; i < size; i++) {
        individuals[i] = genome->clone();
        printf("Cloned individual %d\n", i);
        // hipMalloc(&d_individuals[i], sizeof(CUDAGenome *));
        // printf("Allocated individual %d on device\n", i);
    }
    // hipMemcpy(d_individuals, individuals, size * sizeof(CUDAGenome *), hipMemcpyHostToDevice);
    // printf("Copied individuals' reference from host to device\n");
}

void CUDAPopulation::initialize() {
    printf("Starting initialization\n");
    if (!initialized) {
        for (unsigned int i = 0; i < size; i++) {
            individuals[i]->initialize();
            printf("Initialized individual %d on host\n", i);
        }
        initialized = true;
    }
}

__device__ void CUDAPopulation::step() {
    // Create a temporary population.
    CUDAGenome *ind = (CUDAGenome *) malloc(sizeof(CUDAGenome));
    memcpy(ind, individuals[blockIdx.x], sizeof(CUDAGenome));

    // Select.
    CUDAGenome *partner = select();
    __syncthreads();

    // Crossover.
    if (threadIdx.x == 0) {
        offspring[blockIdx.x] = (CUDAGenome *) malloc(sizeof(CUDAGenome *));
    }
    __syncthreads();
    individuals[blockIdx.x]->crossover(partner, offspring[blockIdx.x]);

    // Mutate.
    offspring[blockIdx.x]->mutate();
    __syncthreads();

    // Overwrite the old individual with the new one.
    if (threadIdx.x == 0) {
        individuals[blockIdx.x] = offspring[blockIdx.x];
    }

    if (blockIdx.x == 0 && threadIdx.x == 0) {
        // Copy the best from the old pop to the new one.
        // TODO.
    }
}

__device__ CUDAGenome *CUDAPopulation::select() {
    float totalFitness = 0.0;
    float previousProb = 0.0;

    if (threadIdx.x == 0) {
        scale();
        sort();
    }
    __syncthreads();

    // Threads of the same block select the same genome by generating the same pseudo-random number.
    hiprandState_t state;
    hiprand_init((unsigned long) clock(), blockIdx.x, 0, &state);
    float random = hiprand_uniform(&state);

    // Calculate the total fitness.
    for (unsigned int i = 0; i < size; i++) {
        totalFitness += individuals[i]->getFitness();
    }

    // Calculate the probability for each individual.
    for (unsigned int i = 0; i < size - 1; i++) {
        float prob = previousProb + (individuals[i]->getFitness() / totalFitness);
        if (random < prob) {
            return individuals[i];
        } else {
            previousProb += prob;
        }
    }
    return individuals[size - 1];
}

__device__ void CUDAPopulation::scale() {
    individuals[blockIdx.x]->scale(individuals[size - 1]->getScore());
}

__device__ void CUDAPopulation::sort() {
    int l;
    CUDAGenome *tmp = (CUDAGenome *) malloc(sizeof(CUDAGenome));

    if (size % 2 == 0) {
        l = size / 2;
    } else {
        l = (size / 2) + 1;
    }

    for (int i = 0; i < l; i++) {
        // Even phase.
        if (!(blockIdx.x & 1) && (blockIdx.x < (size - 1))) {
            if (individuals[blockIdx.x]->getFitness() > individuals[blockIdx.x + 1]->getFitness()) {
                CUDAGenome *tmp = individuals[blockIdx.x];
                individuals[blockIdx.x] = individuals[blockIdx.x + 1];
                individuals[blockIdx.x + 1] = tmp;
            }
        }
        __syncthreads();

        // Odd phase.
        if ((blockIdx.x & 1) && (blockIdx.x < (size - 1))) {
            if (individuals[blockIdx.x]->getFitness() > individuals[blockIdx.x + 1]->getFitness()) {
                CUDAGenome *tmp = individuals[blockIdx.x];
                individuals[blockIdx.x] = individuals[blockIdx.x + 1];
                individuals[blockIdx.x + 1] = tmp;
            }
        }
        __syncthreads();
    }
}
