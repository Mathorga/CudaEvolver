#include "hip/hip_runtime.h"
#include "CUDAPopulation.h"

CUDAPopulation::CUDAPopulation(unsigned int popSize, unsigned int genNum, CUDAGenome &genome, int objective = MAXIMIZE) {
    genNumber = genNum;
    currentGen = 0;
    initialized = false;
    size = popSize;
    individuals = (CUDAGenome **) malloc(size * sizeof(CUDAGenome *));
    hipMalloc(&d_individuals, size * sizeof(CUDAGenome *));
    hipMemcpy(d_individuals, individuals, size * sizeof(CUDAGenome *), hipMemcpyHostToDevice);
    for (unsigned int i = 0; i < size; i++) {
        individuals[i] = genome.clone();
        hipMalloc(&d_individuals[i], sizeof(CUDAGenome));
    }
}

void CUDAPopulation::initialize() {
    if (!initialized) {
        for (unsigned int i = 0; i < size; i++) {
            individuals[i]->initialize();
            hipMemcpy(d_individuals[i], individuals[i], sizeof(CUDAGenome), hipMemcpyHostToDevice);
        }
        initialized = true;
    }
}

void CUDAPopulation::evolve() {
    initialize();
    dim3 gridSize(size);
    dim3 blockSize(individuals[0]->xSize(), individuals[0]->ySize(), individuals[0]->zSize());
    for (unsigned int i = 0; i < genNumber; i++) {
        step<<<gridSize, blockSize>>>();
    }
}

__global__ void CUDAPopulation::step() {
    // Evaluate.
    evaluate();
    for () {
        // Select.
        CUDAGenome *parent1 = select();
        CUDAGenome *parent2 = select();
        // Crossover.
        CUDAGenome *child = crossover(parent1, parent2);
        // Mutate.
        child->mutate();
        // Add the child to the new pop.
        // TODO.
    }
    // Copy the best from the old pop to the new one.
    // TODO.
}

__device__ void CUDAPopulation::evaluate() {
    d_individuals[blockIdx.x]->evaluate();
}

__device__ CUDAGenome *CUDAPopulation::select() {
    // TODO.
}

__device__ CUDAGenome *CUDAPopulation::crossover() {
    // TODO.
}
