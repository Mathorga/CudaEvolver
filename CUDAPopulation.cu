#include "hip/hip_runtime.h"
#include "CUDAPopulation.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void evaluate(CUDAPopulation *pop) {
    pop->individuals[blockIdx.x]->evaluate();
    pop->scale();
}

__global__ void sort() {
    if (blockIdx.x == 0) {
        int l;
        CUDAGenome *tmp = (CUDAGenome *) malloc(sizeof(CUDAGenome));

        if (size % 2 == 0) {
            l = size / 2;
        } else {
            l = (size / 2) + 1;
        }

        for (int i = 0; i < l; i++) {
            // Even phase.
            if (!(threadIdx.x & 1) && (threadIdx.x < (size - 1))) {
                if (individuals[threadIdx.x]->getFitness() > individuals[threadIdx.x + 1]->getFitness()) {
                    CUDAGenome *tmp = individuals[threadIdx.x];
                    individuals[threadIdx.x] = individuals[threadIdx.x + 1];
                    individuals[threadIdx.x + 1] = tmp;
                }
            }
            __syncthreads();

            // Odd phase.
            if ((threadIdx.x & 1) && (threadIdx.x < (size - 1))) {
                if (individuals[threadIdx.x]->getFitness() > individuals[threadIdx.x + 1]->getFitness()) {
                    CUDAGenome *tmp = individuals[threadIdx.x];
                    individuals[threadIdx.x] = individuals[threadIdx.x + 1];
                    individuals[threadIdx.x + 1] = tmp;
                }
            }
            __syncthreads();
        }
    }
}

__global__ void step(CUDAPopulation *pop) {
    pop->step();
}

__global__ void outputBest(CUDAPopulation *pop, *string) {
    if (blockIdx.x == 0) {
        // Output the last (best) individual.
        pop->individuals[pop->getSize() - 1]->output(fileName);
    }
}

__global__ void outputWorst(CUDAPopulation *pop, char *string) {
    if (blockIdx.x == 0) {
        // Output the first (worst) individual.
        pop->individuals[0]->output(string);
    }
}



CUDAPopulation::CUDAPopulation(unsigned int popSize, unsigned int genNum, Objective obj) {
    // printf("Starting creation\n");
    genNumber = genNum;
    currentGen = 0;
    initialized = false;
    size = popSize;
    individuals = (CUDAGenome **) malloc(size * sizeof(CUDAGenome *));
    offspring = (CUDAGenome **) malloc(size * sizeof(CUDAGenome *));
}

__device__ void CUDAPopulation::step() {
    // printf("\n");
    // for (unsigned int i = 0; i < 5; i++) {
    //     printf("x:%u\ty:%u\n", ((CUDAPathGenome *) individuals[blockIdx.x])->path[i].x, ((CUDAPathGenome *) individuals[blockIdx.x])->path[i].y);
    // }

    // Create a temporary population.
    CUDAGenome *ind = (CUDAGenome *) malloc(sizeof(CUDAGenome));
    memcpy(ind, individuals[blockIdx.x], sizeof(CUDAGenome));

    // Select.
    // printf("Selection\n");
    CUDAGenome *partner = select();
    __syncthreads();

    // if (threadIdx.x == 0) {
    //     printf("\n");
    //     for (unsigned int i = 0; i < 5; i++) {
    //         printf("x:%u\ty:%u\n", ((CUDAPathGenome *) individuals[blockIdx.x])->path[i].x, ((CUDAPathGenome *) individuals[blockIdx.x])->path[i].y);
    //     }
    // }

    // Crossover.
    // printf("Crossover\n");
    individuals[blockIdx.x]->crossover(partner, &(offspring[blockIdx.x]));
    __syncthreads();

    // Mutate.
    // printf("Mutation\n");
    offspring[blockIdx.x]->mutate();
    __syncthreads();

    // Overwrite the old individual with the new one.
    if (threadIdx.x == 0) {
        individuals[blockIdx.x] = offspring[blockIdx.x];

        // for (unsigned int i = 0; i < ((CUDAPathGenome *) individuals[blockIdx.x])->getChecksNum(); i++) {
        //     printf("x:%u\ty:%u\n", ((CUDAPathGenome *) individuals[blockIdx.x])->path[i].x, ((CUDAPathGenome *) individuals[blockIdx.x])->path[i].y);
        // }
    }

    if (blockIdx.x == 0 && threadIdx.x == 0) {
        // Copy the best from the old pop to the new one.
        // TODO.
    }
    __syncthreads();
}

__device__ CUDAGenome *CUDAPopulation::select() {
    float totalFitness = 0.0;
    float previousProb = 0.0;

    // Threads of the same block select the same genome by generating the same pseudo-random number.
    hiprandState_t state;
    hiprand_init((unsigned long) clock(), blockIdx.x, 0, &state);
    float random = hiprand_uniform(&state);

    // Calculate the total fitness.
    for (unsigned int i = 0; i < size; i++) {
        totalFitness += individuals[i]->getFitness();
    }

    // Calculate the probability for each individual.
    for (unsigned int i = 0; i < size - 1; i++) {
        float prob = previousProb + (individuals[i]->getFitness() / totalFitness);
        if (random < prob) {
            return individuals[i];
        } else {
            previousProb += prob;
        }
    }
    return individuals[size - 1];
}

__device__ void CUDAPopulation::scale() {
    individuals[blockIdx.x]->scale(individuals[size - 1]->getScore());
}
