#include "hip/hip_runtime.h"
#include "CUDAPathGenome.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void createCUDAPathGenome(CUDAGenome **genome, CUDAPathGenome::_Point2D *checks, unsigned int checksNum) {
    if (threadIdx.x == 0) {
        *genome = new CUDAPathGenome(checks, checksNum);
    }
}

__device__ void CUDAPathGenome::initialize() {
    if (threadIdx.x == 0) {
        // printf("Initializing\n");
        hiprandState_t state;
        hiprand_init((unsigned long) clock(), blockIdx.x, threadIdx.x, &state);

        // Create a copy of the checks array.
        _Point2D *checksCopy = (_Point2D *) malloc((checksNumber + 1) * sizeof(_Point2D));
        for (unsigned int i = 0; i < checksNumber; i++) {
            checksCopy[i] = checks[i];
            // printf("x:%d\ty:%d\n", checks[i].x, checks[i].y);
        }
        // printf("Created a copy of checks\n");

        // Randomly initialize path;
        for (unsigned int i = 0; i < checksNumber; i++) {
            int index = hiprand(&state) % (checksNumber - i);
            path[i] = checksCopy[index];
            for (unsigned int j = index; j < checksNumber - i; j++) {
                checksCopy[j] = checksCopy[j + 1];
            }
        }
    }
}

__device__ void CUDAPathGenome::evaluate() {
    // printf("Evaluating individual %d\n", blockIdx.x);
    __shared__ float *tmpDists;
    tmpDists = (float *) malloc(checksNumber * sizeof(float));

    int bSize = blockDim.x / 2;

    // Calculate distances between each check.
    float dx = (float) path[(threadIdx.x + 1) % checksNumber].x - (float) path[threadIdx.x].x;
    float dy = (float) path[(threadIdx.x + 1) % checksNumber].y - (float) path[threadIdx.x].y;
    tmpDists[threadIdx.x] = sqrtf(powf(dx, 2) + powf(dy, 2));
    __syncthreads();

    // Perform reduction to compute the sum of the distances.
    while (bSize > 0) {
        if (threadIdx.x < bSize) {
            tmpDists[threadIdx.x] += tmpDists[threadIdx.x + bSize];
        }
        bSize /= 2;
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        score = tmpDists[0];
        // printf("Individual %d score: %f\n", blockIdx.x, score);
    }
}

__device__ void CUDAPathGenome::crossover(CUDAGenome *partner, CUDAGenome **offspring) {
    CUDAPathGenome *child = (CUDAPathGenome *) (*offspring);
    CUDAPathGenome *mate = (CUDAPathGenome *) partner;
    // printf("\nChild:\n");
    // // mate->print();
    // child->print();

    // _Point2D *tmpPath = (_Point2D *) malloc(checksNumber * sizeof(_Point2D));
    unsigned int midPoint = 0;

    hiprandState_t state;
    hiprand_init((unsigned long) clock(), 0, 0, &state);
    midPoint = hiprand(&state) % (checksNumber - 1);

    // Pick from parent 1.
    if (threadIdx.x <= midPoint) {
        child->path[threadIdx.x] = path[threadIdx.x];
    }
    __syncthreads();

    // Pick from parent 2.
    if (threadIdx.x == 0) {
        for (unsigned int i = midPoint + 1; i < checksNumber; ) {
            for (unsigned int j = 0; j < checksNumber; j++) {
                bool insert = true;
                for (unsigned int k = 0; k <= midPoint; k++) {
                    if (mate->path[j].id == child->path[k].id) {
                        insert = false;
                        break;
                    }
                }
                if (insert) {
                    printf("Inserting index %u to index %u\n", j, i);
                    child->path[i] = mate->path[j];
                    i++;
                }
            }
        }
    }
    __syncthreads();
}

__device__ void CUDAPathGenome::mutate() {
    // TODO.
}

__device__ CUDAGenome *CUDAPathGenome::clone() {
    return new CUDAPathGenome(checks, checksNumber);
}

__device__ void CUDAPathGenome::scale(float baseScore) {
    if (threadIdx.x == 0) {
        fitness = (baseScore - score) + 1;
        // printf("Individual %d ------- fitness:%f\n", blockIdx.x, fitness);
    }
}

__device__ void CUDAPathGenome::print() {
    #ifdef __CUDA_ARCH__

    if (threadIdx.x == 0) {
        for (unsigned int i = 0; i < checksNumber; i++) {
            printf("x:%u\ty:%u\tid:%d\n", path[i].x, path[i].y, path[i].id);
        }
    }

    #else

    for (unsigned int i = 0; i < checksNumber; i++) {
        printf("x:%u\ty:%u\tid:%d\n", path[i].x, path[i].y, path[i].id);
    }

    #endif
};

__device__ void CUDAPathGenome::output(char *string) {
    for (int i = 0; i < COORD_SIZE; i++) {
        memcpy(&(string[threadIdx.x * POINT_SIZE]), &(path[threadIdx.x].x), COORD_SIZE);
        memcpy(&(string[threadIdx.x * POINT_SIZE + COORD_SIZE]), &(path[threadIdx.x].y), COORD_SIZE);
    }
}


__device__ CUDAPathGenome::CUDAPathGenome(_Point2D *checkArray, unsigned int checksNum) : CUDAGenome(checksNum) {
    checksNumber = checksNum;
    checks = (_Point2D *) malloc(checksNum * sizeof(_Point2D));
    path = (_Point2D *) malloc(checksNum * sizeof(_Point2D));
    distances = (float *) malloc(checksNum * sizeof(float));
    for (unsigned int i = 0; i < checksNum; i++) {
        checks[i] = checkArray[i];
        _Point2D newCheck;
        path[i] = newCheck;
        distances[i] = 0.0;
    }
}
