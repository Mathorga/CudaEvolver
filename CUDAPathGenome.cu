#include "hip/hip_runtime.h"
#include "CUDAPathGenome.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

void CUDAPathGenome::initialize() {
    printf("Initializing\n");
    srand(time(NULL));
    // Create a copy of the checks array.
    _2DDot *checksCopy = (_2DDot *) malloc(checksNumber * sizeof(_2DDot));
    for (unsigned int i = 0; i < checksNumber; i++) {
        checksCopy[i] = checks[i];
    }
    printf("Created a copy of checks\n");

    // Randomly initialize path;
    for (unsigned int i = 0; i < checksNumber; i++) {
        int index = rand() % (checksNumber - i);
        path[i] = checksCopy[index];
        for (unsigned int j = index; j < checksNumber - i; j++) {
            checksCopy[j] = checksCopy[j + 1];
        }
    }
    printf("Initialized path on the host\n");

    // Copy the initialized path on the device copy.
    // hipMalloc(&d_checks, checksNumber * sizeof(_2DDot));
    // hipMalloc(&d_path, checksNumber * sizeof(_2DDot));
    // hipMalloc(&d_distances, checksNumber * sizeof(float));
    // hipMemcpy(d_checks, checks, checksNumber * sizeof(_2DDot), hipMemcpyHostToDevice);
    // hipMemcpy(d_path, path, checksNumber * sizeof(_2DDot), hipMemcpyHostToDevice);
    // printf("Copied path on the device\n");
}

__device__ void CUDAPathGenome::evaluate() {
    printf("Evaluating individual %d\n", blockIdx.x);
    __shared__ float *tmpDists;
    tmpDists = (float *) malloc(checksNumber * sizeof(float));

    int bSize = blockDim.x / 2;

    // Calculate distances between each check.
    float dx = (float) path[(threadIdx.x + 1) % checksNumber].x - (float) path[threadIdx.x].x;
    float dy = (float) path[(threadIdx.x + 1) % checksNumber].y - (float) path[threadIdx.x].y;
    tmpDists[threadIdx.x] = sqrtf(powf(dx, 2) + powf(dy, 2));
    __syncthreads();

    // Perform reduction to compute the sum of the distances.
    while (bSize > 0) {
        if (threadIdx.x < bSize) {
            tmpDists[threadIdx.x] += tmpDists[threadIdx.x + bSize];
        }
        bSize /= 2;
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        score = tmpDists[0];
        printf("Individual %d score: %f\n", blockIdx.x, score);
    }
}

__device__ void CUDAPathGenome::crossover(CUDAGenome *partner, CUDAGenome *offspring) {
    CUDAPathGenome *child = (CUDAPathGenome *) offspring;
    CUDAPathGenome *mate = (CUDAPathGenome *) partner;
    _2DDot *tmpPath = (_2DDot *) malloc(checksNumber * sizeof(_2DDot));
    unsigned int midPoint = 0;

    if (threadIdx.x == 0) {
        hiprandState_t state;
        hiprand_init((unsigned long) clock(), blockIdx.x, 0, &state);
        midPoint = hiprand(&state) % (checksNumber - 1);
    }
    __syncthreads();

    // Pick from parent 1.
    if (threadIdx.x <= midPoint) {
        tmpPath[threadIdx.x] = getCheck(threadIdx.x);
    }
    __syncthreads();

    // Pick from parent 2.
    if (threadIdx.x == 0) {
        for (unsigned int i = midPoint + 1; i < checksNumber; ) {
            for (unsigned int j = 0; j < checksNumber; j++) {
                bool insert = true;
                for (unsigned int k = 0; k <= midPoint; k++) {
                    if (mate->getCheck(j).id == tmpPath[k].id) {
                        insert = false;
                        break;
                    }
                }
                if (insert) {
                    tmpPath[i] = mate->getCheck(j);
                    i++;
                }
            }
        }
    }
    __syncthreads();
    child->setPath(tmpPath);
}

__device__ void CUDAPathGenome::mutate() {
    // TODO.
}

__device__ void CUDAPathGenome::scale(float baseScore) {
    if (threadIdx.x == 0) {
        fitness = (baseScore - score) + 1;
    }
}

CUDAGenome *CUDAPathGenome::clone() {
    return new CUDAPathGenome(checks, checksNumber);
}

void CUDAPathGenome::allocateCopySingle(CUDAGenome **deviceIndividual, CUDAGenome **hostIndividual, hipMemcpyKind direction) {
    hipMalloc(deviceIndividual, sizeof(CUDAPathGenome));
    hipMemcpy(*deviceIndividual, *hostIndividual, sizeof(CUDAPathGenome), direction);

    hipMalloc(((CUDAPathGenome *) (*deviceIndividual))->getDeviceChecksAddress(), (*deviceIndividual)->getXSize() * sizeof(_2DDot));
    hipMalloc(((CUDAPathGenome *) (*deviceIndividual))->getDevicePathAddress(), (*deviceIndividual)->getXSize() * sizeof(_2DDot));
    // hipMalloc(((CUDAPathGenome *) (*deviceIndividual))->getDeviceDistancesAddress(), (*deviceIndividual)->getXSize() * sizeof(float));
    hipMemcpy(((CUDAPathGenome *) (*deviceIndividual))->getDeviceChecks(),
               ((CUDAPathGenome *) (*hostIndividual))->getHostChecks(),
               (*hostIndividual)->getXSize() * sizeof(_2DDot),
               hipMemcpyHostToDevice);
    hipMemcpy(((CUDAPathGenome *) (*deviceIndividual))->getDevicePath(),
               ((CUDAPathGenome *) (*hostIndividual))->getHostPath(),
               (*hostIndividual)->getXSize() * sizeof(_2DDot),
               hipMemcpyHostToDevice);
}

void CUDAPathGenome::allocateCopyMultiple(CUDAGenome ***deviceIndividuals, CUDAGenome ***hostIndividuals, unsigned int count, hipMemcpyKind direction) {
    hipMalloc(deviceIndividuals, count * sizeof(CUDAPathGenome *));
    hipMemcpy(*deviceIndividuals, *hostIndividuals, count * sizeof(CUDAPathGenome *), direction);
}

CUDAPathGenome::CUDAPathGenome(_2DDot *checkArray, unsigned int checksNum) : CUDAGenome(checksNum) {
    checksNumber = checksNum;
    checks = (_2DDot *) malloc(checksNum * sizeof(_2DDot));
    path = (_2DDot *) malloc(checksNum * sizeof(_2DDot));
    distances = (float *) malloc(checksNum * sizeof(float));
    for (unsigned int i = 0; i < checksNum; i++) {
        checks[i] = checkArray[i];
        path[i] = checkArray[i];
        distances[i] = 0.0;
    }

    // hipMalloc(&d_checks, checksNum * sizeof(_2DDot));
    // hipMalloc(&d_path, checksNum * sizeof(_2DDot));
    // hipMalloc(&d_distances, checksNum * sizeof(float));
    // hipMemcpy(d_checks, checkArray, checksNum * sizeof(_2DDot), hipMemcpyHostToDevice);
}
