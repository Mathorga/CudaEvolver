#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "../hpc.h"

#define BLOCK_SIZE 32
#define CHECKS_NUM 15
#define MUTATION_RATE 0.1
#define CROSS_RATE 1

#define cudaCheckError() {                                                                                  \
            hipError_t e = hipGetLastError();                                                             \
            if (e != hipSuccess) {                                                                         \
                printf("Cuda failure %s(%d): %d(%s)\n", __FILE__, __LINE__ - 1, e, hipGetErrorString(e));  \
                exit(0);                                                                                    \
            }                                                                                               \
        }

// Translates bidimensional indexes to a monodimensional one.
// |i| is the column index.
// |j| is the row index.
// |n| is the number of columns (length of the rows).
#define IDX(i, j, n) ((i) * (n) + (j))

typedef unsigned char cell_t;

enum CellContent {
    EMPTY = 0,
    CHECK = 1,
    PATH = 2
};

struct Point2D {
    int x = -1;
    int y = -1;
    int id = -1;
};

struct Individual {
    Point2D path[CHECKS_NUM];
    float score = 0.0;
};

// Code taken from Rosettacode:
// https://rosettacode.org/wiki/Bitmap/Bresenham%27s_line_algorithm#C.2B.2B
// Implementing Bresenham’s line drawing algorithm.
void drawLine(cell_t *field, int n, int x0, int y0, int x1, int y1) {
    int x = x0;
    int y = y0;

    int dx = abs(x1 - x0), sx = x0 < x1 ? 1 : -1;
    int dy = abs(y1 - y0), sy = y0 < y1 ? 1 : -1;
    int err = (dx > dy ? dx : -dy) / 2;
    int e2;

    for (;;) {
        if (!(x == x0 && y == y0) && !(x == x1 && y == y1)) {
            field[IDX(x, y, n)] = PATH;
        }
        if (x == x1 && y == y1) {
            break;
        }
        e2 = err;
        if (e2 > -dx) {
            err -= dy;
            x += sx;
        }
        if (e2 < dy) {
            err += dx;
            y += sy;
        }
    }
}

void dump(const cell_t *field, const Point2D *path, unsigned int n, unsigned int checksNum, const char *filename) {
    cell_t *fieldCopy = (cell_t *) malloc(n * n * sizeof(cell_t));
    for (unsigned int x = 0; x < n; x++) {
        for (unsigned int y = 0; y < n; y++) {
            fieldCopy[IDX(x, y, n)] = field[IDX(x, y, n)];
        }
    }

    if (path != NULL) {
        for (unsigned int i = 0; i < checksNum; i++) {
            drawLine(fieldCopy, n, path[i].x, path[i].y, path[(i + 1) % checksNum].x, path[(i + 1) % checksNum].y);
        }
    }
    FILE *out = fopen(filename, "w");
    if (out == NULL) {
        printf("Cannot create \"%s\"\n", filename);
        abort();
    }
    fprintf(out, "P6\n");
    fprintf(out, "%d %d\n", n, n);
    fprintf(out, "255\n");
    for (unsigned int x = 0; x < n; x++) {
        for (unsigned int y = 0; y < n; y++) {
            if (field[IDX((x + 1) % n, y, n)] == CHECK ||
                field[IDX(x, (y + 1) % n, n)] == CHECK ||
                field[IDX((x + 1) % n, (y + 1) % n, n)] == CHECK ||
                field[IDX((x - 1 + n) % n, y, n)] == CHECK ||
                field[IDX(x, (y - 1 + n) % n, n)] == CHECK ||
                field[IDX((x - 1 + n) % n, (y - 1 + n) % n, n)] == CHECK ||
                field[IDX((x + 1) %n, (y - 1 + n) % n, n)] == CHECK ||
                field[IDX((x - 1 + n) %n, (y + 1) % n, n)] == CHECK) {
                fprintf(out, "%c%c%c", 255, 30, 30);
            } else if (field[IDX(x, y, n)] == CHECK) {
                fprintf(out, "%c%c%c", 20, 20, 0);
            } else if (fieldCopy[IDX(x, y, n)] == EMPTY) {
                fprintf(out, "%c%c%c", 20, 20, 20);
            } else if (fieldCopy[IDX(x, y, n)] == PATH) {
                fprintf(out, "%c%c%c", 250, 175, 53);
            } else {
                printf("Unknown cell state (%d) of cell %d-%d", fieldCopy[IDX(x, y, n)], x, y);
                abort();
            }
        }
    }
    fclose(out);
}












void initialize(Individual *pop, unsigned int popSize, Point2D *checks, unsigned int checksNum) {
    Point2D *checksCopy = (Point2D *) malloc(checksNum * sizeof(Point2D));
    for (unsigned int i = 0; i < popSize; i++) {
        for (unsigned int j = 0; j < checksNum; j++) {
            checksCopy[j] = checks[j];
        }
        for (unsigned int j = 0; j < checksNum; j++) {
            int index = rand() % (checksNum - j);
            pop[i].path[j] = checksCopy[index];
            for (unsigned int k = index; k < checksNum - 1; k++) {
                checksCopy[k] = checksCopy[k + 1];
            }
        }
    }
}

__device__ void evaluate(Individual *family, unsigned int checksNum) {
    family[threadIdx.x].score = 0;
    for (unsigned int i = 0; i < checksNum; i++) {
        family[threadIdx.x].score +=
        sqrtf(powf(fabsf(family[threadIdx.x].path[(i + 1) % checksNum].x - family[threadIdx.x].path[i].x), 2) +
              powf(fabsf(family[threadIdx.x].path[(i + 1) % checksNum].y - family[threadIdx.x].path[i].y), 2));
    }
    // printf("Score of Individual %d of Family %d: %f\n", threadIdx.x, blockIdx.x, family[threadIdx.x].score);
}

__device__ int select(Individual *family, hiprandState_t *state) {
    int random = (int) (hiprand_uniform(state) * blockDim.x);
    return (family[threadIdx.x].score < family[random].score) ? threadIdx.x : random;
}

__device__ void mutate(Individual *family, hiprandState_t *state) {
    Point2D tmp[CHECKS_NUM];
    for (int i = 0; i < CHECKS_NUM; i++) {
        if (hiprand_uniform(state) <= MUTATION_RATE) {
            int firstIndex = i;
            int secondIndex = hiprand_uniform(state) * (CHECKS_NUM - 1);
            for (int j = 0; j < CHECKS_NUM; j++) {
                tmp[j] = family[threadIdx.x].path[j];
            }
            family[threadIdx.x].path[firstIndex] = family[threadIdx.x].path[secondIndex];
            family[threadIdx.x].path[secondIndex] = tmp[firstIndex];
        }
    }
}

__global__ void evolve(Individual *pop, unsigned int genNum, unsigned int checksNum) {
    hiprandState_t state;
    extern __shared__ Individual family[];
    // Individual *tmpFamily = &family[blockDim.x];

    // Initialize the random number generator.
    hiprand_init((unsigned long) clock(), blockIdx.x, threadIdx.x, &state);

    // Copy the family to shared memory.
    family[threadIdx.x] = pop[IDX(blockIdx.x, threadIdx.x, blockDim.x)];

    for (unsigned int g = 0; g < genNum; g++) {
        evaluate(family, checksNum);
        __syncthreads();
        // if (g % 10 == 0) {
        //     migrate();
        // }
        select(family, &state);
        __syncthreads();
        // crossover();
        mutate(family, &state);
        __syncthreads();
    }
    pop[IDX(blockIdx.x, threadIdx.x, blockDim.x)] = family[threadIdx.x];
}















int main(int argc, char const *argv[]) {
    Individual *population;
    // float *scores;

    Individual *d_population;
    Individual *d_tmpPop;
    // float *d_scores;

    unsigned int fieldSize = 500;
    unsigned int popSize = 1024;
    unsigned int famNumber = 32;
    unsigned int genNumber = 1000;

    // char fileName[200];
    double startTime = 0.0;
    double endTime = 0.0;
    cell_t *field;
    Point2D *checks;

    if (argc > 4) {
        printf("Usage: %s [fieldSize [popSize [genNumber]]\n", argv[0]);
        return -1;
    }
    if (argc > 1) {
        fieldSize = atoi(argv[1]);
    }
    if (argc > 2) {
        popSize = atoi(argv[3]);
    }
    if (argc > 3) {
        genNumber = atoi(argv[4]);
    }

    // Create a field of checks.
    field = (cell_t *) malloc(fieldSize * fieldSize * sizeof(cell_t));
    checks = (Point2D *) malloc(CHECKS_NUM * sizeof(Point2D));

    for (unsigned int i = 0; i < fieldSize * fieldSize; i++) {
        field[i] = EMPTY;
    }

    srand(time(NULL));
    for (unsigned int i = 0; i < CHECKS_NUM; i++) {
        checks[i].x = (rand() % fieldSize);
        checks[i].y = (rand() % fieldSize);
        checks[i].id = i;
        field[IDX(checks[i].x, checks[i].y, fieldSize)] = true;
    }

    dump(field, NULL, fieldSize, CHECKS_NUM, "field.ppm");

    printf("Field:\n");
    for (unsigned int i = 0; i < CHECKS_NUM; i++) {
        printf("x:%d\ty:%d\n", checks[i].x, checks[i].y);
    }



    const size_t size = popSize * sizeof(Individual);

    dim3 members(popSize / famNumber);
    dim3 families(famNumber);
    size_t sharedMemSize = members.x * sizeof(Individual);

    // Create the host population.
    population = (Individual *) malloc(size);
    // scores = (float *) malloc(popSize);

    // Create the device populations.
    hipMalloc(&d_population, size);
    hipMalloc(&d_tmpPop, size);
    // hipMalloc(&d_scores, popSize * sizeof(float));

    // Initialize the population.
    initialize(population, popSize, checks, CHECKS_NUM);

    // Copy the host population to the device.
    hipMemcpy(d_population, population, size, hipMemcpyHostToDevice);



    // ***Execution.***
    printf("Execution:\n");
    startTime = hpc_gettime();

    evolve<<<families, members, sharedMemSize>>>(d_population, genNumber, CHECKS_NUM);
    hipDeviceSynchronize();

    endTime = hpc_gettime();
    printf("Execution time (s):%f\n\n", endTime - startTime);




    // Copy the device population back to the host.
    hipMemcpy(population, d_population, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < popSize; i++) {
        char fileName[255];
        sprintf(fileName, "Individual%d.ppm", i);
        // dump(field, population[0].path, fieldSize, CHECKS_NUM, fileName);
    }





    return 0;
}
