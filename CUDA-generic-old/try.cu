#include "hip/hip_runtime.h"
#include "try.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void hi() {
    hiprandState_t state;
    hiprand_init((unsigned long) clock(), blockIdx.x, 0, &state);
    unsigned int random = hiprand(&state);
    float r = hiprand_uniform(&state);
    printf("random int:%u\tconverted:%f\tcapped:%f\n", random, (float) random, r);
}
