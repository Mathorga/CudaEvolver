#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ga/ga.h>
#include "PathGenome.h"

// Translates bidimensional indexes to a monodimensional one.
// |i| is the column index.
// |j| is the row index.
// |n| is the number of columns (length of the rows).
#define IDX(i, j, n) ((i) * (n) + (j))

#define POP_SIZE 10
#define GEN_NUMBER 100
#define MUT_PROBABILITY 0.001

#define FIELD_SIZE 512
#define CHECKS_NUMBER 10

typedef unsigned char cell_t;

enum {
    EMPTY = 0,
    CHECK = 1,
    PATH = 2
};

// Code taken from Rosettacode:
// https://rosettacode.org/wiki/Bitmap/Bresenham%27s_line_algorithm#C.2B.2B
// Implementing Bresenham’s line drawing algorithm.
void drawLine(cell_t *field, int n, int x0, int y0, int x1, int y1) {
    int x = x0;
    int y = y0;

    int dx = abs(x1 - x0), sx = x0 < x1 ? 1 : -1;
    int dy = abs(y1 - y0), sy = y0 < y1 ? 1 : -1;
    int err = (dx > dy ? dx : -dy) / 2, e2;

    for(;;){
        field[IDX(x, y, n)] = PATH;
        if (x == x1 && y == y1) {
            break;
        }
        e2 = err;
        if (e2 >-dx) {
            err -= dy;
            x += sx;
        }
        if (e2 < dy) {
            err += dx;
            y += sy;
        }
    }
}

void dump(cell_t *field, const PathGenome::_2DDot *checks, int n, const char *filename) {
    int i;
    int j;

    // drawLine(field, n, 100, 100, 1, 1);
    for (int i = 0; i < CHECKS_NUMBER; i++) {
        drawLine(field, n, checks[i].x, checks[i].y, checks[(i + 1) % CHECKS_NUMBER].x, checks[(i + 1) % CHECKS_NUMBER].y);
    }
    FILE *out = fopen(filename, "w");
    if (out == NULL) {
        printf("Cannot create \"%s\"\n", filename);
        abort();
    }
    fprintf(out, "P6\n");
    fprintf(out, "%d %d\n", n, n);
    fprintf(out, "255\n");
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            if (field[IDX(i, j, n)] == CHECK) {
                fprintf(out, "%c%c%c", 0, 0, 0);
            } else if (field[IDX(i, j, n)] == EMPTY) {
                fprintf(out, "%c%c%c", 255, 255, 255);
            } else if (field[IDX(i, j, n)] == PATH) {
                fprintf(out, "%c%c%c", 255, 0, 0);
            } else {
                printf("Unknown cell state (%d) of cell %d-%d", field[IDX(i, j, n)], i, j);
                abort();
            }
        }
    }
    fclose(out);
}


// |-----------------------------------------------------------------------------------------|
// Fitness function test.
// |-----------------------------------------------------------------------------------------|
float fitness(GAGenome &g) {
    GA1DBinaryStringGenome &genome = (GA1DBinaryStringGenome &)g;

    float score=0.0;
    for (int i = 0; i < genome.length(); i++) {
        // The more 1s are contained in the string, the higher is the fitness.
        // The score is incremented by the value of the current element of the string (0 or 1).
        score += genome.gene(i);
    }
    return score;
}
// |-----------------------------------------------------------------------------------------|



// |-----------------------------------------------------------------------------------------|
// Initializers.
// |-----------------------------------------------------------------------------------------|
void randomInitializer(GAGenome &g) {
    GA1DBinaryStringGenome &genome=(GA1DBinaryStringGenome &)g;

    for (int i = 0; i < genome.size(); i++) {
        genome.gene(i, GARandomBit());
    }
}

void worstCaseInitializer(GAGenome &g) {
    GA1DBinaryStringGenome &genome=(GA1DBinaryStringGenome &)g;

    for (int i = 0; i < genome.size(); i++) {
        genome.gene(i, 0);
    }
}
// |-----------------------------------------------------------------------------------------|


// |-----------------------------------------------------------------------------------------|
// CUDA population evaluator.
// |-----------------------------------------------------------------------------------------|
__global__ void evaluate(GAPopulation &pop) {
    // pop.individual(threadIdx.x).evaluate();
}

void cudaEvaluator(GAPopulation &p) {
    dim3 blockSize(p.size());

    // TODO Allocate device memory for the population and hipMemcpy it.
    // hipMalloc();
    // hipMemcpy();

    evaluate<<<1, blockSize>>>(p);
}
// |-----------------------------------------------------------------------------------------|



// |-----------------------------------------------------------------------------------------|
// Main.
// |-----------------------------------------------------------------------------------------|
int main(int argc, char const *argv[]) {
    // Create a field of checks.
    cell_t *field;
    PathGenome::_2DDot *checks;

    field = (cell_t *) malloc(FIELD_SIZE * FIELD_SIZE * sizeof(cell_t));
    checks = (PathGenome::_2DDot *) malloc(CHECKS_NUMBER * sizeof(PathGenome::_2DDot));

    for (int i = 0; i < FIELD_SIZE * FIELD_SIZE; i++) {
        field[i] = EMPTY;
    }

    srand(time(NULL));
    for (int i = 0; i < CHECKS_NUMBER; i++) {
        checks[i].x = (int) (rand() % FIELD_SIZE);
        checks[i].y = (int) (rand() % FIELD_SIZE);
        field[IDX(checks[i].x, checks[i].y, FIELD_SIZE)] = true;
    }

    dump(field, checks, FIELD_SIZE, "field.ppm");


    // Create a genome.
    // PathGenome genome();
    //
    // genome.initializer(randomInitializer);
    //
    // // Create a population.
    // GAPopulation population(genome);
    // population.evaluator(cudaEvaluator);
    //
    // // Create the genetic algorithm.
    // GASimpleGA ga(population);
    // ga.nGenerations(GEN_NUMBER);
    // ga.pMutation(MUT_PROBABILITY);
    //
    // ga.initialize();
    //
    // GAPopulation tmpPop = ga.population();
    // printf("\nInitial population:\n");
    // for (int i = 0; i < tmpPop.size(); i++) {
    //     printf("Individual %d: ", i);
    //     GA1DBinaryStringGenome& individual = (GA1DBinaryStringGenome&)tmpPop.individual(i);
    //     for (int j = 0; j < individual.length(); j++) {
    //         printf("%d", individual.gene(j));
    //     }
    //     printf("\n");
    // }
    // printf("\nBest: ");
    // GA1DBinaryStringGenome &currentBest = (GA1DBinaryStringGenome &)tmpPop.best();
    // for (int i = 0; i < currentBest.length(); i++) {
    //     printf("%d", currentBest.gene(i));
    // }
    // printf("\n\n");


    // for (int i = 0; i < ga.nGenerations(); i++) {
    //     // getchar();
    //     printf("\n\n\nGENERATION %d\n", ga.generation() + 1);
    //     ga.step();
    //     GAPopulation tmpPop = ga.population();
    //     // Print the population.
    //     printf("\nPopulation:\n");
    //     for (int i = 0; i < tmpPop.size(); i++) {
    //         printf("Individual %d: ", i);
    //         GA1DBinaryStringGenome& individual = (GA1DBinaryStringGenome&)tmpPop.individual(i);
    //         for (int j = 0; j < individual.length(); j++) {
    //             printf("%d", individual.gene(j));
    //         }
    //         printf("\n");
    //     }
    //     printf("\nBest: ");
    //     currentBest = (GA1DBinaryStringGenome &)tmpPop.best();
    //     for (int i = 0; i < currentBest.length(); i++) {
    //         printf("%d", currentBest.gene(i));
    //     }
    //     printf("\tfitness: %f", tmpPop.max());
    //     printf("\n\n");
    //
    //     // Print statistics.
    //     // std::cout << ga.statistics() << std::endl;
    // }
    return 0;
}
// |-----------------------------------------------------------------------------------------|
