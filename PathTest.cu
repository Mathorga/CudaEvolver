#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ga/ga.h>
#include "PathGenome.h"

// Translates bidimensional indexes to a monodimensional one.
// |i| is the column index.
// |j| is the row index.
// |n| is the number of columns (length of the rows).
#define IDX(i, j, n) ((i) * (n) + (j))

#define POP_SIZE 10
#define GEN_NUMBER 100
#define MUT_RATE 0.001

#define FIELD_SIZE 100
#define CHECKS_NUMBER 10

typedef unsigned char cell_t;

enum {
    EMPTY = 0,
    CHECK = 1,
    PATH = 2
};

// Code taken from Rosettacode:
// https://rosettacode.org/wiki/Bitmap/Bresenham%27s_line_algorithm#C.2B.2B
// Implementing Bresenham’s line drawing algorithm.
void drawLine(cell_t *field, int n, int x0, int y0, int x1, int y1) {
    int x = x0;
    int y = y0;

    int dx = abs(x1 - x0), sx = x0 < x1 ? 1 : -1;
    int dy = abs(y1 - y0), sy = y0 < y1 ? 1 : -1;
    int err = (dx > dy ? dx : -dy) / 2, e2;

    for(;;){
        if (!(x == x0 && y == y0) && !(x == x1 && y == y1)) {
            field[IDX(x, y, n)] = PATH;
        }
        if (x == x1 && y == y1) {
            break;
        }
        e2 = err;
        if (e2 >-dx) {
            err -= dy;
            x += sx;
        }
        if (e2 < dy) {
            err += dx;
            y += sy;
        }
    }
}

void dump(const cell_t *field, const PathGenome::_2DDot *path, int n, const char *filename) {

    cell_t *fieldCopy = (cell_t *) malloc(n * n * sizeof(cell_t));
    for (int x = 0; x < n; x++) {
        for (int y = 0; y < n; y++) {
            fieldCopy[IDX(x, y, n)] = field[IDX(x, y, n)];
        }
    }

    if (path != NULL) {
        for (int i = 0; i < CHECKS_NUMBER; i++) {
            drawLine(fieldCopy, n, path[i].x, path[i].y, path[(i + 1) % CHECKS_NUMBER].x, path[(i + 1) % CHECKS_NUMBER].y);
        }
    }
    FILE *out = fopen(filename, "w");
    if (out == NULL) {
        printf("Cannot create \"%s\"\n", filename);
        abort();
    }
    fprintf(out, "P6\n");
    fprintf(out, "%d %d\n", n, n);
    fprintf(out, "255\n");
    for (int x = 0; x < n; x++) {
        for (int y = 0; y < n; y++) {
            if (fieldCopy[IDX((x + 1) % n, y, n)] == CHECK ||
                fieldCopy[IDX(x, (y + 1) % n, n)] == CHECK ||
                fieldCopy[IDX((x + 1) % n, (y + 1) % n, n)] == CHECK ||
                fieldCopy[IDX((x - 1 + n) % n, y, n)] == CHECK ||
                fieldCopy[IDX(x, (y - 1 + n) % n, n)] == CHECK ||
                fieldCopy[IDX((x - 1 + n) % n, (y - 1 + n) % n, n)] == CHECK ||
                fieldCopy[IDX((x + 1) %n, (y - 1 + n) % n, n)] == CHECK ||
                fieldCopy[IDX((x - 1 + n) %n, (y + 1) % n, n)] == CHECK) {
                fprintf(out, "%c%c%c", 255, 30, 30);
            } else if (fieldCopy[IDX(x, y, n)] == CHECK) {
                fprintf(out, "%c%c%c", 20, 20, 0);
            } else if (fieldCopy[IDX(x, y, n)] == EMPTY) {
                fprintf(out, "%c%c%c", 20, 20, 20);
            } else if (fieldCopy[IDX(x, y, n)] == PATH) {
                fprintf(out, "%c%c%c", 250, 175, 53);
            } else {
                printf("Unknown cell state (%d) of cell %d-%d", fieldCopy[IDX(x, y, n)], x, y);
                abort();
            }
        }
    }
    fclose(out);
}




float fitness(GAGenome &g) {
    GA1DBinaryStringGenome &genome = (GA1DBinaryStringGenome &)g;

    float score=0.0;
    for (int i = 0; i < genome.length(); i++) {
        // The more 1s are contained in the string, the higher is the fitness.
        // The score is incremented by the value of the current element of the string (0 or 1).
        score += genome.gene(i);
    }
    return score;
}





void cudaEvaluator(GAPopulation &p) {
    dim3 blockSize(CHECKS_NUMBER);
    for (int i = 0; i < p.size(); i++) {
        GAGenome *individual = &(p.individual(i));
        // Allocate memory for the genome object on the device.
        GAGenome *d_individual;
        hipMalloc(&d_individual, sizeof(GAGenome));
        // Copy the genome object to the device.
        hipMemcpy(d_individual, &individual, sizeof(GAGenome), hipMemcpyHostToDevice);

        // Allocate memory for the genome object's pointers on the device.
        // PathGenome::_2DDot *d_checks;
        PathGenome *ind = (PathGenome *) individual;
        PathGenome::_2DDot *d_path;
        float *d_distances;
        // hipMalloc(&d_checks, sizeof(PathGenome::_2DDot));
        hipMalloc(&d_path, sizeof(PathGenome::_2DDot));
        hipMalloc(&d_distances, sizeof(float));
        // Copy the genome object' pointers on the device.
        hipMemcpy(d_path, ind->getPath(), sizeof(PathGenome::_2DDot), hipMemcpyHostToDevice);
        hipMemcpy(d_distances, ind->getDistances(), sizeof(float), hipMemcpyHostToDevice);

        // evaluate(d_individual);

        // Set the score.
    }
}




int main(int argc, char const *argv[]) {
    // Create a field of checks.
    cell_t *field;
    PathGenome::_2DDot *checks;

    field = (cell_t *) malloc(FIELD_SIZE * FIELD_SIZE * sizeof(cell_t));
    checks = (PathGenome::_2DDot *) malloc(CHECKS_NUMBER * sizeof(PathGenome::_2DDot));

    for (int i = 0; i < FIELD_SIZE * FIELD_SIZE; i++) {
        field[i] = EMPTY;
    }

    srand(time(NULL));
    for (int i = 0; i < CHECKS_NUMBER; i++) {
        checks[i].x = (rand() % FIELD_SIZE);
        checks[i].y = (rand() % FIELD_SIZE);
        checks[i].id = i;
        field[IDX(checks[i].x, checks[i].y, FIELD_SIZE)] = true;
    }

    dump(field, NULL, FIELD_SIZE, "field.ppm");

    std::cout << "Field:\n";
    for (int i = 0; i < CHECKS_NUMBER; i++) {
        std::cout << "x:" << checks[i].x << "\ty:" << checks[i].y << "\n";
    }

    // Create a genome.
    PathGenome genome(CHECKS_NUMBER, checks);

    // for (int i = 0; i < 1000; i++) {
    //     genome.mutate(MUT_RATE);
    //     char fileName[200];
    //     snprintf(fileName, 200, "pathMut%d.ppm", i);
    //     dump(field, genome.getPath(), FIELD_SIZE, fileName);
    // }

    // for (int i = 0; i < 100; i++) {
    //     genome.initialize();
    //     char fileName[200];
    //     snprintf(fileName, 200, "path%d.ppm", i);
    //     dump(field, genome.getPath(), FIELD_SIZE, fileName);
    // }

    // Create a population.
    GAPopulation population(genome, POP_SIZE);
    population.initialize();
    population.evaluator(cudaEvaluator);
    // cudaEvaluator(population);
    std::cout << "\nPopulation\n" << population;

    population.evaluate();
    for (int i = 0; i < population.size(); i++) {
        std::cout << "individual " << i << " - score: " << ((PathGenome &) (population.individual(i))).score() << "\n";
    }

    // // Create the genetic algorithm.
    // GASimpleGA ga(population);
    // ga.nGenerations(GEN_NUMBER);
    // ga.pMutation(MUT_RATE);
    //
    // ga.initialize();
    //
    // GAPopulation tmpPop = ga.population();
    // printf("\nInitial population:\n");
    // for (int i = 0; i < tmpPop.size(); i++) {
    //     printf("Individual %d: ", i);
    //     GA1DBinaryStringGenome& individual = (GA1DBinaryStringGenome&)tmpPop.individual(i);
    //     for (int j = 0; j < individual.length(); j++) {
    //         printf("%d", individual.gene(j));
    //     }
    //     printf("\n");
    // }
    // printf("\nBest: ");
    // GA1DBinaryStringGenome &currentBest = (GA1DBinaryStringGenome &)tmpPop.best();
    // for (int i = 0; i < currentBest.length(); i++) {
    //     printf("%d", currentBest.gene(i));
    // }
    // printf("\n\n");


    // for (int i = 0; i < ga.nGenerations(); i++) {
    //     // getchar();
    //     printf("\n\n\nGENERATION %d\n", ga.generation() + 1);
    //     ga.step();
    //     GAPopulation tmpPop = ga.population();
    //     // Print the population.
    //     printf("\nPopulation:\n");
    //     for (int i = 0; i < tmpPop.size(); i++) {
    //         printf("Individual %d: ", i);
    //         GA1DBinaryStringGenome& individual = (GA1DBinaryStringGenome&)tmpPop.individual(i);
    //         for (int j = 0; j < individual.length(); j++) {
    //             printf("%d", individual.gene(j));
    //         }
    //         printf("\n");
    //     }
    //     printf("\nBest: ");
    //     currentBest = (GA1DBinaryStringGenome &)tmpPop.best();
    //     for (int i = 0; i < currentBest.length(); i++) {
    //         printf("%d", currentBest.gene(i));
    //     }
    //     printf("\tfitness: %f", tmpPop.max());
    //     printf("\n\n");
    //
    //     // Print statistics.
    //     // std::cout << ga.statistics() << std::endl;
    // }

    // delete a;
    return 0;
}
