#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <ga/ga.h>
#include "PathGenome.h"

// Translates bidimensional indexes to a monodimensional one.
// |i| is the column index.
// |j| is the row index.
// |n| is the number of columns (length of the rows).
#define IDX(i, j, n) ((i) * (n) + (j))

#define POP_SIZE 5
#define GEN_NUMBER 300
#define MUT_RATE 0.1

#define FIELD_SIZE 50
#define CHECKS_NUMBER 5

typedef unsigned char cell_t;

enum {
    EMPTY = 0,
    CHECK = 1,
    PATH = 2
};

// Code taken from Rosettacode:
// https://rosettacode.org/wiki/Bitmap/Bresenham%27s_line_algorithm#C.2B.2B
// Implementing Bresenham’s line drawing algorithm.
void drawLine(cell_t *field, int n, int x0, int y0, int x1, int y1) {
    int x = x0;
    int y = y0;

    int dx = abs(x1 - x0), sx = x0 < x1 ? 1 : -1;
    int dy = abs(y1 - y0), sy = y0 < y1 ? 1 : -1;
    int err = (dx > dy ? dx : -dy) / 2;
    int e2;

    for(;;){
        if (!(x == x0 && y == y0) && !(x == x1 && y == y1)) {
            field[IDX(x, y, n)] = PATH;
        }
        if (x == x1 && y == y1) {
            break;
        }
        e2 = err;
        if (e2 > -dx) {
            err -= dy;
            x += sx;
        }
        if (e2 < dy) {
            err += dx;
            y += sy;
        }
    }
}

void dump(const cell_t *field, const PathGenome::_2DDot *path, int n, const char *filename) {

    cell_t *fieldCopy = (cell_t *) malloc(n * n * sizeof(cell_t));
    for (int x = 0; x < n; x++) {
        for (int y = 0; y < n; y++) {
            fieldCopy[IDX(x, y, n)] = field[IDX(x, y, n)];
        }
    }

    if (path != NULL) {
        for (int i = 0; i < CHECKS_NUMBER; i++) {
            drawLine(fieldCopy, n, path[i].x, path[i].y, path[(i + 1) % CHECKS_NUMBER].x, path[(i + 1) % CHECKS_NUMBER].y);
        }
    }
    FILE *out = fopen(filename, "w");
    if (out == NULL) {
        printf("Cannot create \"%s\"\n", filename);
        abort();
    }
    fprintf(out, "P6\n");
    fprintf(out, "%d %d\n", n, n);
    fprintf(out, "255\n");
    for (int x = 0; x < n; x++) {
        for (int y = 0; y < n; y++) {
            if (field[IDX((x + 1) % n, y, n)] == CHECK ||
                field[IDX(x, (y + 1) % n, n)] == CHECK ||
                field[IDX((x + 1) % n, (y + 1) % n, n)] == CHECK ||
                field[IDX((x - 1 + n) % n, y, n)] == CHECK ||
                field[IDX(x, (y - 1 + n) % n, n)] == CHECK ||
                field[IDX((x - 1 + n) % n, (y - 1 + n) % n, n)] == CHECK ||
                field[IDX((x + 1) %n, (y - 1 + n) % n, n)] == CHECK ||
                field[IDX((x - 1 + n) %n, (y + 1) % n, n)] == CHECK) {
                fprintf(out, "%c%c%c", 255, 30, 30);
            } else if (field[IDX(x, y, n)] == CHECK) {
                fprintf(out, "%c%c%c", 20, 20, 0);
            } else if (fieldCopy[IDX(x, y, n)] == EMPTY) {
                fprintf(out, "%c%c%c", 20, 20, 20);
            } else if (fieldCopy[IDX(x, y, n)] == PATH) {
                fprintf(out, "%c%c%c", 250, 175, 53);
            } else {
                printf("Unknown cell state (%d) of cell %d-%d", fieldCopy[IDX(x, y, n)], x, y);
                abort();
            }
        }
    }
    fclose(out);
}




float fitness(GAGenome &g) {
    GA1DBinaryStringGenome &genome = (GA1DBinaryStringGenome &)g;

    float score=0.0;
    for (int i = 0; i < genome.length(); i++) {
        // The more 1s are contained in the string, the higher is the fitness.
        // The score is incremented by the value of the current element of the string (0 or 1).
        score += genome.gene(i);
    }
    return score;
}




void cudaEvaluator(GAPopulation &p) {
    dim3 blockSize(CHECKS_NUMBER);
    for (int i = 0; i < p.size(); i++) {
        GAGenome *individual = &(p.individual(i));
        // Allocate memory for the genome object on the device.
        GAGenome *d_individual;
        hipMalloc(&d_individual, sizeof(GAGenome));
        // Copy the genome object to the device.
        hipMemcpy(d_individual, &individual, sizeof(GAGenome), hipMemcpyHostToDevice);

        // Allocate memory for the genome object's pointers on the device.
        // PathGenome::_2DDot *d_checks;
        PathGenome *ind = (PathGenome *) individual;
        PathGenome::_2DDot *d_path;
        float *d_distances;
        // hipMalloc(&d_checks, sizeof(PathGenome::_2DDot));
        hipMalloc(&d_path, sizeof(PathGenome::_2DDot));
        hipMalloc(&d_distances, sizeof(float));
        // Copy the genome object' pointers on the device.
        hipMemcpy(d_path, ind->getPath(), sizeof(PathGenome::_2DDot), hipMemcpyHostToDevice);
        hipMemcpy(d_distances, ind->getDistances(), sizeof(float), hipMemcpyHostToDevice);

        // Set the score.
    }
}



__global__ void kernel() {
    printf("%d\n", threadIdx.x);
}


int main(int argc, char const *argv[]) {
    // Create a field of checks.
    kernel<<<1, 10>>>();
    cell_t *field;
    PathGenome::_2DDot *checks;

    field = (cell_t *) malloc(FIELD_SIZE * FIELD_SIZE * sizeof(cell_t));
    checks = (PathGenome::_2DDot *) malloc(CHECKS_NUMBER * sizeof(PathGenome::_2DDot));

    for (int i = 0; i < FIELD_SIZE * FIELD_SIZE; i++) {
        field[i] = EMPTY;
    }

    srand(time(NULL));
    for (int i = 0; i < CHECKS_NUMBER; i++) {
        checks[i].x = (rand() % FIELD_SIZE);
        checks[i].y = (rand() % FIELD_SIZE);
        checks[i].id = i;
        field[IDX(checks[i].x, checks[i].y, FIELD_SIZE)] = true;
    }

    dump(field, NULL, FIELD_SIZE, "field.ppm");

    std::cout << "Field:\n";
    for (int i = 0; i < CHECKS_NUMBER; i++) {
        std::cout << "x:" << checks[i].x << "\ty:" << checks[i].y << "\n";
    }

    // Create a genome.
    PathGenome genome(CHECKS_NUMBER, checks);

    // for (int i = 0; i < 100; i++) {
    //     genome.initialize();
    //     char fileName[200];
    //     snprintf(fileName, 200, "path%d.ppm", i);
    //     dump(field, genome.getPath(), FIELD_SIZE, fileName);
    // }

    // Create a population.
    GAPopulation population(genome, POP_SIZE);
    // population.initialize();
    // population.evaluator(cudaEvaluator);
    // cudaEvaluator(population);

    // Create the genetic algorithm.
    GASimpleGA ga(population);
    ga.nGenerations(GEN_NUMBER);
    ga.pMutation(MUT_RATE);

    ga.initialize();
    ga.minimize();
    ((PathGenome &) ga.population().individual(0)).evaluate();

    // ga.evolve();
    // std::cout << "\n1st population\n" << ga.population();

    // printf("\nInitial population:\n");
    // for (int i = 0; i < ga.population().size(); i++) {
    //     printf("Individual %d:\n", i);
    //     PathGenome &individual = (PathGenome &)ga.population().individual(i);
    //     for (unsigned int j = 0; j < individual.getChecksNum(); j++) {
    //         printf("x:%d\ty:%d\n", individual.gene(j).x, individual.gene(j).y);
    //     }
    //     printf("\n");
    //     printf("hello");
    // }
    // printf("\nBest: ");
    // GA1DBinaryStringGenome &currentBest = (GA1DBinaryStringGenome &)tmpPop.best();
    // for (int i = 0; i < currentBest.length(); i++) {
    //     printf("%d", currentBest.gene(i));
    // }
    // printf("\n\n");


    for (int i = 0; i < ga.nGenerations(); i++) {
        getchar();
        printf("\n\n\nGENERATION %d\n", ga.generation() + 1);
        ga.step();
        printf("\nbest score:%f\n", ga.population().max());
        GAPopulation tmpPop = ga.population();
        // Print the population.
        // printf("\nPopulation:\n");
        // for (int i = 0; i < tmpPop.size(); i++) {
        //     printf("Individual %d: ", i);
        //     GA1DBinaryStringGenome& individual = (GA1DBinaryStringGenome&)tmpPop.individual(i);
        //     for (int j = 0; j < individual.length(); j++) {
        //         printf("%d", individual.gene(j));
        //     }
        //     printf("\n");
        // }
        // printf("\nBest: ");
        // currentBest = (GA1DBinaryStringGenome &) tmpPop.best();
        // for (int i = 0; i < currentBest.length(); i++) {
        //     printf("%d", currentBest.gene(i));
        // }
        char fileName[200];
        snprintf(fileName, 200, "BestOfGen%d.ppm", i);
        dump(field, ((PathGenome &) tmpPop.best()).getPath(), FIELD_SIZE, fileName);

        printf("\n\n");

        // Print statistics.
        // std::cout << ga.statistics() << std::endl;
    }

    // delete a;
    return 0;
}
